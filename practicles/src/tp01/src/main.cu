
#include <hip/hip_runtime.h>
#include <string>
#include <fstream>
#include <iostream>
#include <sstream>

__device__
int mod(int a, int b) {
    return ((a%=b)<0)?a+b:a;
}

__global__
void caesarEncode(int N, char* buffer, char* outBuffer, int shiftValue) {
	int stride = blockDim.x * gridDim.x;
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	for (int i = index; i<N; i += stride) {
		if ('a'<=buffer[i] && buffer[i]<='z' || 'A'<=buffer[i] && buffer[i]<='Z') {
			outBuffer[i] = buffer[i]<='Z'?
				mod((buffer[i]-'A')+shiftValue, 26)+'A':
				mod((buffer[i]-'a')+shiftValue, 26)+'a';
		} else {
			outBuffer[i] = buffer[i];
		}
	}
}

bool testInvariant() {
	bool test_ok = true;
	int N = 32;

	char *buffer, *outBuffer;
	hipMallocManaged(&buffer, N*sizeof(char));
	hipMallocManaged(&outBuffer, N*sizeof(char));

	strcpy(buffer, "abcdefghijklmnopqrstuvwxyzABCDZ");

	caesarEncode<<<1, 32>>>(N, buffer, outBuffer, 3);
	hipDeviceSynchronize();

	if (strcmp(outBuffer, "defghijklmnopqrstuvwxyzabcDEFGC")!=0) {
		test_ok = false;
	}

	caesarEncode<<<1, 32>>>(N, outBuffer, outBuffer, -3);
	hipDeviceSynchronize();

	if (strcmp(buffer, outBuffer)!=0) {
		test_ok = false;
	}

	std::cout<<"testInvariant: "<<(test_ok?"pass":"fail")<<std::endl;

	hipFree(buffer);
	hipFree(outBuffer);

	return test_ok;
}

bool testNonAlphabeticalChar() {
	bool test_ok = true;
	int N = 32;

	char *buffer;
	hipMallocManaged(&buffer, N*sizeof(char));

	strcpy(buffer, "aAzZ1234[]/*-+$*! .,| HIJ%42 ��");

	caesarEncode<<<1, 32>>>(N, buffer, buffer, 1);
	hipDeviceSynchronize();

	if (strcmp(buffer, "bBaA1234[]/*-+$*! .,| IJK%42 ��")!=0) {
		test_ok = false;
	}

	std::cout<<"testNonAlphabeticalChar: "<<(test_ok?"pass":"fail")<<std::endl;

	hipFree(buffer);

	return test_ok;
}

int main(int argc, char *argv[])
{
	testInvariant();
	testNonAlphabeticalChar();

	if (argc<2) {
		std::cout<<"No file specified."<<std::endl;
		return 0;
	}

	std::ifstream file(argv[1], std::ios::in);
	std::string str(static_cast<std::stringstream const&>(std::stringstream()<<file.rdbuf()).str());

	char *buffer;
	hipMallocManaged(&buffer, str.size()*sizeof(char));

	memcpy(buffer, str.c_str(), str.size());

	caesarEncode<<<32, 256>>>(str.size(), buffer, buffer, 3);

	hipDeviceSynchronize();

	std::ofstream fileout(std::string(argv[1]) + std::string(".out.txt"), std::ios::out);
	fileout.write(buffer, str.size());

	hipFree(buffer);

	return 0;
}
